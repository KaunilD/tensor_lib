#include "device_tensor.hpp"


struct cuda_deletor {
	void operator()(float* p) const {
		hipFree(p);
	}
};

template<int DIMS>
void device_tensor<DIMS>::allocate_data() {
	hipMalloc(&(this->m_data), this->get_n_elems() * sizeof(float));
	this->m_data_ptr = std::shared_ptr<float>(this->m_data, cuda_deletor());
};

template <int DIMS>
device_tensor<DIMS>::device_tensor(const std::array<int, DIMS> t_size) :tensor<DIMS>(t_size) {
	allocate_data();
};

template class device_tensor<1>;