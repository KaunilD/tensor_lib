﻿#include "tensor.hpp"

template<int DIMS>
__host__ __device__
float* tensor<DIMS>::get() const {
	return m_data;
};

template<int DIMS>
void tensor<DIMS>::set_n_elems() {
	m_num_elements = 1;
	for (int i = 0; i < m_size.size(); i++) {
		m_num_elements *= m_size[i];
	}
};

template<int DIMS>
tensor<DIMS>::tensor(const std::array<int, DIMS> t_size) : m_size(t_size) {
	static_assert(DIMS <= 3, "Tensor class only supports upto 3 dimensions.\n");
	static_assert(DIMS >= 1, "Tensor must be atleast 1 dimensional!\n");
	set_n_elems();
};

template<int DIMS>
__host__ __device__
size_t tensor<DIMS>::get_n_elems() const {
	return m_num_elements;
};

template<int DIMS>
__host__ __device__
float& tensor<DIMS>::at(size_t x) const {
	static_assert(DIMS == 1, "Trying to use 1D accessor on a non 1D-Tensor.\n");
	return *(this->m_data + x);
};

template<int DIMS>
__host__ __device__
float& tensor<DIMS>::at(size_t x, size_t y) const {
	static_assert(DIMS == 2, "Trying to use 2D accessor on a non 2D-Tensor.\n");
	return *(this->m_data + x * m_size[1] + y);
};

template<int DIMS>
__host__ __device__
float& tensor<DIMS>::at(size_t x, size_t y, size_t z) const {
	static_assert(DIMS == 3, "Trying to use 3D accessor on a non 3D-Tensor.\n");
	return *(this->m_data + x * m_size[1] * m_size[2] + y * m_size[2] + z);
};

template class tensor<1>;
template class tensor<2>;
template class tensor<3>;
