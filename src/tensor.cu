﻿#include "tensor.hpp"

template<int DIMS>
__host__ __device__ 
float* tensor<DIMS>::get() const { 
	return m_data; 
};


template<int DIMS>
void tensor<DIMS>::set_n_elems() {
	m_num_elements += m_size[0];
};

template<int DIMS>
tensor<DIMS>::tensor(const std::array<int, DIMS> t_size) : m_size(t_size) {
	static_assert(DIMS == 1, "Only 1D Tensors supported!");
	set_n_elems();
};

template<int DIMS>
__host__ __device__ 
uint32_t tensor<DIMS>::get_n_elems() const {
	return m_num_elements; 
};

template<int DIMS>
__host__ __device__ 
float& tensor<DIMS>::at(size_t x) {
	return *(this->m_data + x);
};

template class tensor<1>;